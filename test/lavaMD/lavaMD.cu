#include "hip/hip_runtime.h"
//========================================================================================================================================================================================================200
//======================================================================================================================================================150
//====================================================================================================100
//==================================================50

//========================================================================================================================================================================================================200
//	UPDATE
//========================================================================================================================================================================================================200

//	14 APR 2011 Lukasz G. Szafaryn

//========================================================================================================================================================================================================200
//	DEFINE/INCLUDE
//========================================================================================================================================================================================================200

//======================================================================================================================================================150
//	LIBRARIES
//======================================================================================================================================================150

#include <stdio.h>					// (in path known to compiler)			needed by printf
#include <stdlib.h>					// (in path known to compiler)			needed by malloc
#include <stdbool.h>				// (in path known to compiler)			needed by true/false

//======================================================================================================================================================150
//	UTILITIES
//======================================================================================================================================================150
#include <sys/time.h>

#ifdef __cplusplus
extern "C" {
#endif

//===============================================================================================================================================================================================================200
//	TIMER CODE
//===============================================================================================================================================================================================================200

//======================================================================================================================================================150

//======================================================================================================================================================150
//	FUNCTIONS
//======================================================================================================================================================150

//====================================================================================================100
//	DISPLAY TIME
//====================================================================================================100

 // Returns the current system time in microseconds
long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}

//===============================================================================================================================================================================================================200
//	END TIMER CODE
//===============================================================================================================================================================================================================200

#ifdef __cplusplus
}
#endif

#ifdef __cplusplus
extern "C" {
#endif

//===============================================================================================================================================================================================================200
//	DESCRIPTION
//===============================================================================================================================================================================================================200

// Returns:	0 if string does not represent integer
//			1 if string represents integer

//===============================================================================================================================================================================================================200
//	NUM CODE
//===============================================================================================================================================================================================================200

//======================================================================================================================================================150
//	ISINTEGER FUNCTION
//======================================================================================================================================================150

int isInteger(char *str){

	//====================================================================================================100
	//	make sure it's not empty
	//====================================================================================================100

	if (*str == '\0'){
		return 0;
	}

	//====================================================================================================100
	//	if any digit is not a number, return false
	//====================================================================================================100

	for(; *str != '\0'; str++){
		if (*str < 48 || *str > 57){	// digit characters (need to include . if checking for float)
			return 0;
		}
	}

	//====================================================================================================100
	//	it got past all my checks so I think it's a number
	//====================================================================================================100

	return 1;
}

//===============================================================================================================================================================================================================200
//	END NUM CODE
//===============================================================================================================================================================================================================200

#ifdef __cplusplus
}
#endif


//======================================================================================================================================================150
//	MAIN FUNCTION HEADER
//======================================================================================================================================================150

//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200
//	DEFINE / INCLUDE
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200

#define fp double

#define NUMBER_PAR_PER_BOX 100							// keep this low to allow more blocks that share shared memory to run concurrently, code does not work for larger than 110, more speedup can be achieved with larger number and no shared memory used

/* #define NUMBER_THREADS 128								// this should be roughly equal to NUMBER_PAR_PER_BOX for best performance */

// Parameterized work group size
#ifdef RD_WG_SIZE_0_0
        #define NUMBER_THREADS RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define NUMBER_THREADS RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define NUMBER_THREADS RD_WG_SIZE
#else
        #define NUMBER_THREADS 128
#endif


#define DOT(A,B) ((A.x)*(B.x)+(A.y)*(B.y)+(A.z)*(B.z))	// STABLE

//===============================================================================================================================================================================================================200
//	STRUCTURES
//===============================================================================================================================================================================================================200

typedef struct
{
	fp x, y, z;

} THREE_VECTOR;

typedef struct
{
	fp v, x, y, z;

} FOUR_VECTOR;

typedef struct nei_str
{

	// neighbor box
	int x, y, z;
	int number;
	long offset;

} nei_str;

typedef struct box_str
{

	// home box
	int x, y, z;
	int number;
	long offset;

	// neighbor boxes
	int nn;
	nei_str nei[26];

} box_str;

typedef struct par_str
{

	fp alpha;

} par_str;

typedef struct dim_str
{

	// input arguments
	int cur_arg;
	int arch_arg;
	int cores_arg;
	int boxes1d_arg;

	// system memory
	long number_boxes;
	long box_mem;
	long space_elem;
	long space_mem;
	long space_mem2;

} dim_str;

//===============================================================================================================================================================================================================200
//	FUNCTION PROTOTYPES
//===============================================================================================================================================================================================================200

int 
main(	int argc, 
		char *argv []);

//======================================================================================================================================================150
//	KERNEL
//======================================================================================================================================================150

void setdevice(void){

	// variables
	int num_devices;
	int device;

	// work
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
		
		// variables
		int max_multiprocessors; 
		int max_device;
		hipDeviceProp_t properties;

		// initialize variables
		max_multiprocessors = 0;
		max_device = 0;
		
		for (device = 0; device < num_devices; device++) {
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}

}

//====================================================================================================100
//	GET LAST ERROR
//====================================================================================================100

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		// fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		printf("Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		fflush(NULL);
		exit(EXIT_FAILURE);
	}
}	
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200
//	plasmaKernel_gpu_2
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200
struct SharedSpace {
	FOUR_VECTOR rA_shared[100];
	FOUR_VECTOR rB_shared[100];
	double qB_shared[100];
};

__launch_bounds__(NUMBER_THREADS)
__global__ void kernel_gpu_cuda(par_str d_par_gpu,
								dim_str d_dim_gpu,
								box_str* d_box_gpu,
								FOUR_VECTOR* d_rv_gpu,
								fp* d_qv_gpu,
#ifdef USE_OSM
								SharedSpace* osm,
#endif
								FOUR_VECTOR* d_fv_gpu)
{

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	THREAD PARAMETERS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	int bx = blockIdx.x;																// get current horizontal block index (0-n)
	int tx = threadIdx.x;															// get current horizontal thread index (0-n)
	// int ax = bx*NUMBER_THREADS+tx;
	// int wbx = bx;
	int wtx = tx;

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	DO FOR THE NUMBER OF BOXES
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	if(bx<d_dim_gpu.number_boxes){
	// while(wbx<box_indexes_counter){

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Extract input parameters
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// parameters
		fp a2 = 2.0*d_par_gpu.alpha*d_par_gpu.alpha;

		// home box
		int first_i;
		FOUR_VECTOR* rA;
		FOUR_VECTOR* fA;

		// nei box
		int pointer;
		int k = 0;
		int first_j;
		FOUR_VECTOR* rB;
		fp* qB;
		int j = 0;

#ifdef USE_OSM
		auto* rA_shared = osm[blockIdx.x].rA_shared;
		auto* rB_shared = osm[blockIdx.x].rB_shared;
		auto* qB_shared = osm[blockIdx.x].qB_shared;
#else
	__shared__ SharedSpace osm;
#define rA_shared osm.rA_shared
#define rB_shared osm.rB_shared
#define qB_shared osm.qB_shared
#endif

		// common
		fp r2;
		fp u2;
		fp vij;
		fp fs;
		fp fxij;
		fp fyij;
		fp fzij;
		THREE_VECTOR d;

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Home box
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Setup parameters
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - box parameters
		first_i = d_box_gpu[bx].offset;

		// home box - distance, force, charge and type parameters
		rA = &d_rv_gpu[first_i];
		fA = &d_fv_gpu[first_i];

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Copy to shared memory
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - shared memory
		while(wtx<NUMBER_PAR_PER_BOX){
			rA_shared[wtx] = rA[wtx];
			wtx = wtx + NUMBER_THREADS;
		}
		wtx = tx;

		// synchronize threads  - not needed, but just to be safe
		__syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// loop over neiing boxes of home box
		for (k=0; k<(1+d_box_gpu[bx].nn); k++){

			//----------------------------------------50
			//	nei box - get pointer to the right box
			//----------------------------------------50

			if(k==0){
				pointer = bx;													// set first box to be processed to home box
			}
			else{
				pointer = d_box_gpu[bx].nei[k-1].number;							// remaining boxes are nei boxes
			}

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - box parameters
			first_j = d_box_gpu[pointer].offset;

			// nei box - distance, (force), charge and (type) parameters
			rB = &d_rv_gpu[first_j];
			qB = &d_qv_gpu[first_j];

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - shared memory
			while(wtx<NUMBER_PAR_PER_BOX){
				rB_shared[wtx] = rB[wtx];
				qB_shared[wtx] = qB[wtx];
				wtx = wtx + NUMBER_THREADS;
			}
			wtx = tx;

			// synchronize threads because in next section each thread accesses data brought in by different threads here
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation
			//----------------------------------------------------------------------------------------------------------------------------------140

			// loop for the number of particles in the home box
			// for (int i=0; i<nTotal_i; i++){
			while(wtx<NUMBER_PAR_PER_BOX){

				// loop for the number of particles in the current nei box
				for (j=0; j<NUMBER_PAR_PER_BOX; j++){

					// r2 = rA[wtx].v + rB[j].v - DOT(rA[wtx],rB[j]); 
					// u2 = a2*r2;
					// vij= exp(-u2);
					// fs = 2.*vij;

					// d.x = rA[wtx].x  - rB[j].x;
					// fxij=fs*d.x;
					// d.y = rA[wtx].y  - rB[j].y;
					// fyij=fs*d.y;
					// d.z = rA[wtx].z  - rB[j].z;
					// fzij=fs*d.z;

					// fA[wtx].v +=  qB[j]*vij;
					// fA[wtx].x +=  qB[j]*fxij;
					// fA[wtx].y +=  qB[j]*fyij;
					// fA[wtx].z +=  qB[j]*fzij;



					r2 = (fp)rA_shared[wtx].v + (fp)rB_shared[j].v - DOT((fp)rA_shared[wtx],(fp)rB_shared[j]); 
					u2 = a2*r2;
					vij= exp(-u2);
					fs = 2*vij;

					d.x = (fp)rA_shared[wtx].x  - (fp)rB_shared[j].x;
					fxij=fs*d.x;
					d.y = (fp)rA_shared[wtx].y  - (fp)rB_shared[j].y;
					fyij=fs*d.y;
					d.z = (fp)rA_shared[wtx].z  - (fp)rB_shared[j].z;
					fzij=fs*d.z;

					fA[wtx].v +=  (double)((fp)qB_shared[j]*vij);
					fA[wtx].x +=  (double)((fp)qB_shared[j]*fxij);
					fA[wtx].y +=  (double)((fp)qB_shared[j]*fyij);
					fA[wtx].z +=  (double)((fp)qB_shared[j]*fzij);

				}

				// increment work thread index
				wtx = wtx + NUMBER_THREADS;

			}

			// reset work index
			wtx = tx;

			// synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation END
			//----------------------------------------------------------------------------------------------------------------------------------140

		}

		// // increment work block index
		// wbx = wbx + NUMBER_BLOCKS;

		// // synchronize - because next iteration will overwrite current shared memory
		// __syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop END
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

	}

}

void 
kernel_gpu_cuda_wrapper(par_str par_cpu,
						dim_str dim_cpu,
						box_str* box_cpu,
						FOUR_VECTOR* rv_cpu,
						fp* qv_cpu,
						FOUR_VECTOR* fv_cpu)
{

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	// timer
	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;

	time0 = get_time();

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	hipDeviceSynchronize();

	//====================================================================================================100
	//	VARIABLES
	//====================================================================================================100

	box_str* d_box_gpu;
	FOUR_VECTOR* d_rv_gpu;
	fp* d_qv_gpu;
	FOUR_VECTOR* d_fv_gpu;

	dim3 threads;
	dim3 blocks;

	//====================================================================================================100
	//	EXECUTION PARAMETERS
	//====================================================================================================100

	blocks.x = dim_cpu.number_boxes;
	blocks.y = 1;
	threads.x = NUMBER_THREADS;											// define the number of threads in the block
	threads.y = 1;

	time1 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY				(MALLOC)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	boxes
	//==================================================50

	hipMalloc(	(void **)&d_box_gpu, 
				dim_cpu.box_mem);

	//==================================================50
	//	rv
	//==================================================50

	hipMalloc(	(void **)&d_rv_gpu, 
				dim_cpu.space_mem);

	//==================================================50
	//	qv
	//==================================================50

	hipMalloc(	(void **)&d_qv_gpu, 
				dim_cpu.space_mem2);

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY
	//====================================================================================================100

	//==================================================50
	//	fv
	//==================================================50

	hipMalloc(	(void **)&d_fv_gpu, 
				dim_cpu.space_mem);

#ifdef USE_OSM
        SharedSpace* osm;
        hipMalloc((void**)&osm, sizeof(SharedSpace) * blocks.x);
#endif

    time2 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	boxes
	//==================================================50

	hipMemcpy(	d_box_gpu, 
				box_cpu,
				dim_cpu.box_mem, 
				hipMemcpyHostToDevice);

	//==================================================50
	//	rv
	//==================================================50

	hipMemcpy(	d_rv_gpu,
				rv_cpu,
				dim_cpu.space_mem,
				hipMemcpyHostToDevice);

	//==================================================50
	//	qv
	//==================================================50

	hipMemcpy(	d_qv_gpu,
				qv_cpu,
				dim_cpu.space_mem2,
				hipMemcpyHostToDevice);

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY
	//====================================================================================================100

	//==================================================50
	//	fv
	//==================================================50
	hipMemcpy(	d_fv_gpu, 
				fv_cpu, 
				dim_cpu.space_mem, 
				hipMemcpyHostToDevice);

	time3 = get_time();

	//======================================================================================================================================================150
	//	KERNEL
	//======================================================================================================================================================150

	// launch kernel - all boxes
	kernel_gpu_cuda<<<blocks, threads>>>(	par_cpu,
											dim_cpu,
											d_box_gpu,
											d_rv_gpu,
											d_qv_gpu,
#ifdef USE_OSM
											osm,
#endif
											d_fv_gpu);

	checkCUDAError("Start");
	hipDeviceSynchronize();

	time4 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY (CONTD.)
	//======================================================================================================================================================150

	hipMemcpy(	fv_cpu, 
				d_fv_gpu, 
				dim_cpu.space_mem, 
				hipMemcpyDeviceToHost);

	time5 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY DEALLOCATION
	//======================================================================================================================================================150
#ifdef USE_OSM
	hipFree(osm);
#endif
	hipFree(d_rv_gpu);
	hipFree(d_qv_gpu);
	hipFree(d_fv_gpu);
	hipFree(d_box_gpu);

	time6 = get_time();

	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150

	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time6-time0) * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) (time6-time0) / 1000000);

}

//========================================================================================================================================================================================================200
//	MAIN FUNCTION
//========================================================================================================================================================================================================200

int 
main(	int argc, 
		char *argv [])
{

	printf("thread block size of kernel = %d \n", NUMBER_THREADS);
	//======================================================================================================================================================150
	//	CPU/MCPU VARIABLES
	//======================================================================================================================================================150

	// timer
	long long time0;

	time0 = get_time();

	// timer
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;
	long long time7;

	// counters
	int i, j, k, l, m, n;

	// system memory
	par_str par_cpu;
	dim_str dim_cpu;
	box_str* box_cpu;
	FOUR_VECTOR* rv_cpu;
	fp* qv_cpu;
	FOUR_VECTOR* fv_cpu;
	int nh;

	time1 = get_time();

	//======================================================================================================================================================150
	//	CHECK INPUT ARGUMENTS
	//======================================================================================================================================================150

	// assing default values
	dim_cpu.boxes1d_arg = 1;

	// go through arguments
	for(dim_cpu.cur_arg=1; dim_cpu.cur_arg<argc; dim_cpu.cur_arg++){
		// check if -boxes1d
		if(strcmp(argv[dim_cpu.cur_arg], "-boxes1d")==0){
			// check if value provided
			if(argc>=dim_cpu.cur_arg+1){
				// check if value is a number
				if(isInteger(argv[dim_cpu.cur_arg+1])==1){
					dim_cpu.boxes1d_arg = atoi(argv[dim_cpu.cur_arg+1]);
					if(dim_cpu.boxes1d_arg<0){
						printf("ERROR: Wrong value to -boxes1d parameter, cannot be <=0\n");
						return 0;
					}
					dim_cpu.cur_arg = dim_cpu.cur_arg+1;
				}
				// value is not a number
				else{
					printf("ERROR: Value to -boxes1d parameter in not a number\n");
					return 0;
				}
			}
			// value not provided
			else{
				printf("ERROR: Missing value to -boxes1d parameter\n");
				return 0;
			}
		}
		// unknown
		else{
			printf("ERROR: Unknown parameter\n");
			return 0;
		}
	}

	// Print configuration
	printf("Configuration used: boxes1d = %d\n", dim_cpu.boxes1d_arg);

	time2 = get_time();

	//======================================================================================================================================================150
	//	INPUTS
	//======================================================================================================================================================150

	par_cpu.alpha = 0.5;

	time3 = get_time();

	//======================================================================================================================================================150
	//	DIMENSIONS
	//======================================================================================================================================================150

	// total number of boxes
	dim_cpu.number_boxes = dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg;

	// how many particles space has in each direction
	dim_cpu.space_elem = dim_cpu.number_boxes * NUMBER_PAR_PER_BOX;
	dim_cpu.space_mem = dim_cpu.space_elem * sizeof(FOUR_VECTOR);
	dim_cpu.space_mem2 = dim_cpu.space_elem * sizeof(fp);

	// box array
	dim_cpu.box_mem = dim_cpu.number_boxes * sizeof(box_str);

	time4 = get_time();

	//======================================================================================================================================================150
	//	SYSTEM MEMORY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	BOX
	//====================================================================================================100

	// allocate boxes
	box_cpu = (box_str*)malloc(dim_cpu.box_mem);

	// initialize number of home boxes
	nh = 0;

	// home boxes in z direction
	for(i=0; i<dim_cpu.boxes1d_arg; i++){
		// home boxes in y direction
		for(j=0; j<dim_cpu.boxes1d_arg; j++){
			// home boxes in x direction
			for(k=0; k<dim_cpu.boxes1d_arg; k++){

				// current home box
				box_cpu[nh].x = k;
				box_cpu[nh].y = j;
				box_cpu[nh].z = i;
				box_cpu[nh].number = nh;
				box_cpu[nh].offset = nh * NUMBER_PAR_PER_BOX;

				// initialize number of neighbor boxes
				box_cpu[nh].nn = 0;

				// neighbor boxes in z direction
				for(l=-1; l<2; l++){
					// neighbor boxes in y direction
					for(m=-1; m<2; m++){
						// neighbor boxes in x direction
						for(n=-1; n<2; n++){

							// check if (this neighbor exists) and (it is not the same as home box)
							if(		(((i+l)>=0 && (j+m)>=0 && (k+n)>=0)==true && ((i+l)<dim_cpu.boxes1d_arg && (j+m)<dim_cpu.boxes1d_arg && (k+n)<dim_cpu.boxes1d_arg)==true)	&&
									(l==0 && m==0 && n==0)==false	){

								// current neighbor box
								box_cpu[nh].nei[box_cpu[nh].nn].x = (k+n);
								box_cpu[nh].nei[box_cpu[nh].nn].y = (j+m);
								box_cpu[nh].nei[box_cpu[nh].nn].z = (i+l);
								box_cpu[nh].nei[box_cpu[nh].nn].number =	(box_cpu[nh].nei[box_cpu[nh].nn].z * dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg) + 
																			(box_cpu[nh].nei[box_cpu[nh].nn].y * dim_cpu.boxes1d_arg) + 
																			 box_cpu[nh].nei[box_cpu[nh].nn].x;
								box_cpu[nh].nei[box_cpu[nh].nn].offset = box_cpu[nh].nei[box_cpu[nh].nn].number * NUMBER_PAR_PER_BOX;

								// increment neighbor box
								box_cpu[nh].nn = box_cpu[nh].nn + 1;

							}

						} // neighbor boxes in x direction
					} // neighbor boxes in y direction
				} // neighbor boxes in z direction

				// increment home box
				nh = nh + 1;

			} // home boxes in x direction
		} // home boxes in y direction
	} // home boxes in z direction

	//====================================================================================================100
	//	PARAMETERS, DISTANCE, CHARGE AND FORCE
	//====================================================================================================100

	// random generator seed set to random value - time in this case
	srand(time(NULL));

	// input (distances)
	rv_cpu = (FOUR_VECTOR*)malloc(dim_cpu.space_mem);
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		rv_cpu[i].v = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
		rv_cpu[i].x = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
		rv_cpu[i].y = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
		rv_cpu[i].z = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
	}

	// input (charge)
	qv_cpu = (fp*)malloc(dim_cpu.space_mem2);
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		qv_cpu[i] = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
	}

	// output (forces)
	fv_cpu = (FOUR_VECTOR*)malloc(dim_cpu.space_mem);
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		fv_cpu[i].v = 0;								// set to 0, because kernels keeps adding to initial value
		fv_cpu[i].x = 0;								// set to 0, because kernels keeps adding to initial value
		fv_cpu[i].y = 0;								// set to 0, because kernels keeps adding to initial value
		fv_cpu[i].z = 0;								// set to 0, because kernels keeps adding to initial value
	}

	time5 = get_time();

	//======================================================================================================================================================150
	//	KERNEL
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU_CUDA
	//====================================================================================================100

	kernel_gpu_cuda_wrapper(par_cpu,
							dim_cpu,
							box_cpu,
							rv_cpu,
							qv_cpu,
							fv_cpu);

	time6 = get_time();

	//======================================================================================================================================================150
	//	SYSTEM MEMORY DEALLOCATION
	//======================================================================================================================================================150

	// dump results
#ifdef OUTPUT
        FILE *fptr;
	fptr = fopen("result.txt", "w");	
	for(i=0; i<dim_cpu.space_elem; i=i+1){
        	fprintf(fptr, "%f, %f, %f, %f\n", fv_cpu[i].v, fv_cpu[i].x, fv_cpu[i].y, fv_cpu[i].z);
	}
	fclose(fptr);
#endif       	



	free(rv_cpu);
	free(qv_cpu);
	free(fv_cpu);
	free(box_cpu);

	time7 = get_time();

	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150

	// printf("Time spent in different stages of the application:\n");

	// printf("%15.12f s, %15.12f % : VARIABLES\n",						(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time7-time0) * 100);
	// printf("%15.12f s, %15.12f % : INPUT ARGUMENTS\n", 					(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time7-time0) * 100);
	// printf("%15.12f s, %15.12f % : INPUTS\n",							(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time7-time0) * 100);
	// printf("%15.12f s, %15.12f % : dim_cpu\n", 							(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time7-time0) * 100);
	// printf("%15.12f s, %15.12f % : SYS MEM: ALO\n",						(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time7-time0) * 100);

	// printf("%15.12f s, %15.12f % : KERNEL: COMPUTE\n",					(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time7-time0) * 100);

	// printf("%15.12f s, %15.12f % : SYS MEM: FRE\n", 					(float) (time7-time6) / 1000000, (float) (time7-time6) / (float) (time7-time0) * 100);

	// printf("Total time:\n");
	// printf("%.12f s\n", 												(float) (time7-time0) / 1000000);

	//======================================================================================================================================================150
	//	RETURN
	//======================================================================================================================================================150

	return 0.0;																					// always returns 0.0

}
